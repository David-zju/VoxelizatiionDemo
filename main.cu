#include "hip/hip_runtime.h"
#include "geom.h"

auto add_midpoint(vector<double> &arr) {
    vector<double> ret;
    for (int i = 0, n = arr.size(); i < n - 1; i ++) {
        ret.push_back(arr[i]);
        ret.push_back((arr[i] + arr[i + 1]) / 2);
    }
    ret.push_back(arr.back());
    return ret;
}

__global__ void kernel_get_faces_in_cells(
    buffer<double3> verts, buffer<int4> faces,
    buffer<double> xs, buffer<double> ys, int dir,
    buffer<int> len, buffer<int4> out) {
    for (int n = cuIdx(x); n < faces.len; n += cuDim(x)) {
        auto face = faces[n];
        auto a = verts[face.x], b = verts[face.y], c = verts[face.z];
        auto p0 = rotate(fmin(a, b, c), dir), p1 = rotate(fmax(a, b, c), dir);
        for (int i = 0, nx = xs.len; i < nx - 1; i ++) {
            auto x0 = xs[i], x1 = xs[i + 1];
            for (int j = 0, ny = ys.len; j < ny - 1; j ++) {
                auto y0 = ys[j], y1 = ys[j + 1];
                if (p0.x < x1 && p0.y < y1 && p1.x > x0 && p1.y > y0) {
                    auto next = atomicAdd(len.ptr + i + j * nx, 1);
                    if (next < out.len) {
                        out[next] = face;
                    }
                }
            }
        }
    }
}

struct face_groups_t {
    vector<double> xs, ys;
    vector<int> offset;
    vector<int4> faces;
};

auto get_faces_in_cells(device_vector<double3> &verts, device_vector<int4> &faces, grid_t &grid, int dir) {
    auto &xv = dir == 0 ? grid.ys : dir == 1 ? grid.zs : grid.xs;
    auto &yv = dir == 0 ? grid.zs : dir == 1 ? grid.xs : grid.ys;
    face_groups_t ret { xv, yv };

    device_vector xs(xv), ys(yv);
    ret.offset.resize(xs.len * ys.len + 1);
    device_vector<int> offset(ret.offset);
    kernel_get_faces_in_cells CU_DIM(1024, 128) (verts, faces, xs, ys, dir, offset, { });
    CUDA_ASSERT(hipGetLastError());

    offset.copy_to(ret.offset);
    auto sum = accumulate(ret.offset.begin(), ret.offset.end(), 0);
    device_vector<int4> out(sum);
    auto offset_vec = ret.offset;
    exclusive_scan(offset_vec.begin(), offset_vec.end(), ret.offset.begin(), 0);

    offset.copy_from(ret.offset);
    kernel_get_faces_in_cells CU_DIM(1024, 128) (verts, faces, xs, ys, dir, offset, out);
    CUDA_ASSERT(hipGetLastError());

    out.copy_to(ret.faces);
    return ret;
}

__host__ __device__ __forceinline__ auto lerp(double a, double b, double f) {
    return a * (1. - f) + b * f;
}
__host__ __device__ __forceinline__ auto interp(double2 a, double2 b, double x) {
    auto f = (x - a.x) / (b.x - a.x);
    return lerp(a.y, b.y, f);
}
__host__ __device__ __forceinline__ auto interp(double3 a, double3 b, double x) {
    auto f = (x - a.x) / (b.x - a.x);
    return double2 { lerp(a.y, b.y, f), lerp(a.z, b.z, f) };
}
__host__ __device__ __forceinline__ auto ordered(double x0, double x, double x1) {
    return (x0 < x1 && x0 <= x && x <= x1) || (x1 < x0 && x1 <= x && x <= x0);
}
__host__ __device__ __forceinline__ auto point_in_triangle(double2 p, double3 A, double3 B, double3 C, int dir) {
    double2 u = { 0 }, v = { 0 };
    auto a = rotate(A, dir), b = rotate(B, dir), c = rotate(C, dir);
    auto AB = ordered(a.x, p.x, b.x),
         BC = ordered(b.x, p.x, c.x),
         CA = ordered(c.x, p.x, a.x);
    if (CA && AB) {
        u = interp(a, b, p.x); v = interp(a, c, p.x);
    } else if (AB && BC) {
        u = interp(b, c, p.x); v = interp(b, a, p.x);
    } else if (BC && CA) {
        u = interp(c, a, p.x); v = interp(c, b, p.x);
    }
    if (ordered(u.x, p.y, v.x)) {
        return interp(u, v, p.y);
    }
    return DBL_MAX;
}

struct cast_joint_t {
    short solid;
    float pos;
};
auto operator<(const cast_joint_t &a, const cast_joint_t &b) {
    return a.solid == b.solid ? a.pos < b.pos : a.solid < b.solid;
}
__global__ void kernel_cast_in_cells(
    buffer<double3> verts, buffer<int4> faces,
    buffer<double> xs, buffer<double> ys, int dir,
    buffer<int> offset, double tol,
    buffer<int> len, buffer<cast_joint_t> out) {
    int u = blockIdx.x, v = blockIdx.y, w = u + gridDim.x * v;
    if (u >= xs.len - 1 || v >= ys.len - 1) {
        return;
    }
    int i = threadIdx.x, j = threadIdx.y, k = i + blockDim.x * j;
    double2 p = {
        lerp(xs[u] + tol, xs[u + 1] - tol, 1. * i / (blockDim.x - 1)),
        lerp(ys[v] + tol, ys[v + 1] - tol, 1. * j / (blockDim.y - 1)),
    };
    for (int m0 = offset[w], m1 = offset[w + 1]; m0 < m1; m0 ++) {
        auto face = faces[m0];
        auto pos = point_in_triangle(p, verts[face.x], verts[face.y], verts[face.z], dir);
        if (pos != DBL_MAX) {
            auto next = atomicAdd(len.ptr + w * blockDim.x * blockDim.y + k, 1);
            if (next < out.len) {
                out[next] = { (short) face.w, (float) pos };
            }
        }
    }
}

auto dump_gltf(vector<double3> &vx, vector<double3> &vy, vector<double3> &vz, string file, int mode) {
    vector<float3> out;
    for (auto v : vx) {
        auto p = revert(v, 0);
        out.push_back({ (float) p.x, (float) p.y, (float) p.z });
    }
    for (auto v : vy) {
        auto p = revert(v, 1);
        out.push_back({ (float) p.x, (float) p.y, (float) p.z });
    }
    for (auto v : vz) {
        auto p = revert(v, 2);
        out.push_back({ (float) p.x, (float) p.y, (float) p.z });
    }
    float3 p0 = { FLT_MAX, FLT_MAX, FLT_MAX }, p1 = { FLT_MIN, FLT_MIN, FLT_MIN };
    if (out.size()) {
        p0 = out.back(); p1 = out.front();
        for (auto p : out) {
            p0.x = fmin(p0.x, p.x); p1.x = fmax(p1.x, p.x);
            p0.y = fmin(p0.y, p.y); p1.y = fmax(p1.y, p.y);
            p0.z = fmin(p0.z, p.z); p1.z = fmax(p1.z, p.z);
        }
        if (mode == 1) {
            auto d = float3 { p1.x - p0.x, p1.y - p0.y, p1.z - p0.z },
                 c = float3 { p1.x + p0.x, p1.y + p0.y, p1.z + p0.z };
            out.push_back({ c.x / 2, c.y / 2, p0.z - d.z });
            out.push_back({ c.x / 2, c.y / 2, p1.z + d.z });
            out.push_back({ c.x / 2, p0.y - d.y, c.z / 2 });
            out.push_back({ c.x / 2, p1.y + d.z, c.z / 2 });
            out.push_back({ p0.x - d.x, c.y / 2, c.z / 2 });
            out.push_back({ p1.x + d.x, c.y / 2, c.z / 2 });
            for (int i = out.size() - 6; i < out.size(); i ++) {
                auto p = out[i];
                p0.x = fmin(p0.x, p.x); p1.x = fmax(p1.x, p.x);
                p0.y = fmin(p0.y, p.y); p1.y = fmax(p1.y, p.y);
                p0.z = fmin(p0.z, p.z); p1.z = fmax(p1.z, p.z);
            }
        }
    }
    std::ofstream fn(file + ".bin", std::ios::out | std::ios::binary);
    auto byteLength = out.size() * sizeof(float3);
    fn.write((char *) out.data(), byteLength);

    json j;
    std::ifstream("tool/view.gltf") >> j;
    for (auto &item : j["meshes"]) {
        for (auto &prim : item["primitives"]) {
            prim["mode"] = mode;
        }
    }
    for (auto &item : j["accessors"]) {
        item["min"][0] = p0.x; item["min"][1] = p0.y; item["min"][2] = p0.z;
        item["max"][0] = p1.x; item["max"][1] = p1.y; item["max"][2] = p1.z;
        item["count"] = out.size();
    }
    for (auto &item : j["bufferViews"]) {
        item["byteLength"] = byteLength;
    }
    auto filename = filesystem::path(file).filename().u8string();
    for (auto &item : j["buffers"]) {
        item["byteLength"] = byteLength;
        item["uri"] = filename + ".bin";
    }
    std::ofstream(file) << j.dump(2);
}

struct cast_dexel_t {
    double tol;
    int pixels;
    vector<double> xs, ys;
    vector<int> offset;
    vector<cast_joint_t> joints;
    auto &sort_joints() {
        auto ptr = joints.data();
        for (int m = 0, n = offset.size(); m < n - 1; m ++) {
            auto begin = offset[m], end = offset[m + 1];
            sort(ptr + begin, ptr + end);
        }
        return *this;
    }
    auto get_verts(int mode) {
        vector<double3> verts;
        for (int u = 0; u + 1 < xs.size(); u ++) for (int v = 0; v + 1 < ys.size(); v ++) {
            int w = u + v * xs.size();
            for (int i = 0; i < pixels; i ++) for (int j = 0; j < pixels; j ++) {
                int k = i + j * pixels;
                double2 p = {
                    lerp(xs[u] + tol, xs[u + 1] - tol, 1. * i / (pixels - 1)),
                    lerp(ys[v] + tol, ys[v + 1] - tol, 1. * j / (pixels - 1)),
                };
                auto m = w * pixels * pixels + k;
                auto begin = offset[m], end = offset[m + 1];
                if (mode == 0) {
                    for (int q = begin; q < end; q ++) {
                        auto &a = joints[q];
                        verts.push_back({ p.x, p.y, a.pos });
                    }
                } else {
                    for (int q = begin; q < end - 1; q ++) {
                        auto &a = joints[q], &b = joints[q + 1];
                        if (a.solid == b.solid) {
                            verts.push_back({ p.x, p.y, a.pos + tol });
                            verts.push_back({ p.x, p.y, b.pos - tol });
                            q ++;
                        }
                    }
                }
            }
        }
        return verts;
    }
    auto dump_gltf(string file, int dir, int mode) {
        vector<double3> vx, vy, vz;
        if (dir == 0) {
            vx = get_verts(mode);
        } else if (dir == 1) {
            vy = get_verts(mode);
        } else if (dir == 2) {
            vz = get_verts(mode);
        }
        ::dump_gltf(vx, vy, vz, file, mode);
    }
};
auto cast_in_cells(device_vector<double3> &verts, face_groups_t &groups, int dir, int pixels, double tol) {
    device_vector xs(groups.xs), ys(groups.ys);
    device_vector faces(groups.faces);
    device_vector offset(groups.offset);
    dim3 gridDim((int) xs.len, (int) ys.len, 1),
         blockDim(pixels, pixels, 1);

    cast_dexel_t ret { tol, pixels, groups.xs, groups.ys };
    ret.offset.resize(xs.len * ys.len * blockDim.x * blockDim.y + 1);
    device_vector len(ret.offset);
    kernel_cast_in_cells CU_DIM(gridDim, blockDim) (verts, faces, xs, ys, dir, offset, tol, len, { });
    CUDA_ASSERT(hipGetLastError());

    len.copy_to(ret.offset);
    auto sum = accumulate(ret.offset.begin(), ret.offset.end(), 0);
    device_vector<cast_joint_t> out(sum);
    auto offset_vec = ret.offset;
    exclusive_scan(offset_vec.begin(), offset_vec.end(), ret.offset.begin(), 0);

    len.copy_from(ret.offset);
    kernel_cast_in_cells CU_DIM(gridDim, blockDim) (verts, faces, xs, ys, dir, offset, tol, len, out);
    CUDA_ASSERT(hipGetLastError());

    out.copy_to(ret.joints);
    return ret;
}

int main() {
    grid_t grid;
    grid.from_json("data\\MaterialAndGridLines.json");
    grid.xs = add_midpoint(grid.xs); //grid.xs = add_midpoint(grid.xs);
    grid.ys = add_midpoint(grid.ys); //grid.ys = add_midpoint(grid.ys);
    grid.zs = add_midpoint(grid.zs); //grid.zs = add_midpoint(grid.zs);
    auto nx = grid.xs.size(), ny = grid.ys.size(), nz = grid.zs.size();
    printf("INFO: loaded %zu x %zu x %zu (%zu) grids\n", nx, ny, nz, nx * ny * nz);

    mesh_t mesh;
    mesh.from_obj("data\\toStudent_EM2.obj");
    if (0) {
        int geom = 52;
        printf("WARN: filter mesh only to show g == %d\n", geom);
        auto mesh_faces = mesh.faces;
        mesh.faces.resize(0);
        for (auto face : mesh_faces) {
            if (face.w == geom) {
                mesh.faces.push_back(face);
            }
        }
    }
    printf("INFO: loaded %zu faces with %zu vertices\n", mesh.faces.size(), mesh.verts.size());

    if (0) {
        grid.xs = { -2, -1, 0, 1, 2 };
        grid.ys = { -2, -1, 0, 1, 2 };
        grid.zs = { -2, -1, 0, 1, 2 };
        mesh.verts = {
            { 0, 0, 0 },
            { 0.5, 0, 0 },
            { 0, 0.7, 0 },
            { 0, 0, 0.9 },
        };
        mesh.faces = {
            { 0, 1, 2, 0 },
            { 0, 1, 3, 0 },
            { 0, 2, 3, 0 },
            { 1, 2, 3, 0 },
        };
    }

    // WARN: it seems impossible to dump more pixels
    int mode = 0, pixels = 32;
    double tol = 1e-3;

    device_vector verts(mesh.verts);
    device_vector faces(mesh.faces);
    auto all_start = clock_now();
    vector<double3> dexels[3];
    for (int dir = 0; dir < 3; dir ++) {
        auto axis = ("xyz")[dir];
        auto start_group = clock_now();
        auto groups = get_faces_in_cells(verts, faces, grid, dir);
        printf("PERF: on %c group %zu in %f s\n", axis, groups.faces.size(), seconds_since(start_group));
        auto start_cast = clock_now();
        auto casted = cast_in_cells(verts, groups, dir, pixels, tol);
        printf("PERF: on %c cast %zu in %f s\n", axis, casted.joints.size(), seconds_since(start_cast));
        //dexels[dir] = casted.sort_joints().get_verts(mode);
    }
    dump_gltf(dexels[0], dexels[1], dexels[2], "data/dump.gltf", mode);
    printf("PERF: all done in %f s\n", seconds_since(all_start));

    return 0;
}
