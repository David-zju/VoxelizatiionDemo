#include "hip/hip_runtime.h"
#include "geom.h"

auto add_midpoint(vector<double> &arr) {
    vector<double> ret;
    for (int i = 0, n = arr.size(); i < n - 1; i ++) {
        ret.push_back(arr[i]);
        ret.push_back((arr[i] + arr[i + 1]) / 2);
    }
    ret.push_back(arr.back());
    return ret;
}

__global__ void kernel_get_faces_in_cells(
    buffer<double3> verts, buffer<int4> faces,
    buffer<double> xs, buffer<double> ys, int dir,
    buffer<int> len, buffer<int4> out) {
    for (int n = cuIdx(x); n < faces.len; n += cuDim(x)) {
        auto face = faces[n];
        auto x = verts[face.x], y = verts[face.y], z = verts[face.z];
        auto p0 = rotate(fmin(x, y, z), dir), p1 = rotate(fmax(x, y, z), dir);
        for (int i = 0, nx = xs.len; i < nx - 1; i ++) {
            auto x0 = xs[i], x1 = xs[i + 1];
            for (int j = 0, ny = ys.len; j < ny - 1; j ++) {
                auto y0 = ys[j], y1 = ys[j + 1];
                if (p0.x < x1 && p0.y < y1 &&
                    p1.x > x0 && p1.y > y0) {
                    auto next = atomicAdd(len.ptr + i + j * nx, 1);
                    if (next < out.len) {
                        out[next] = face;
                    }
                }
            }
        }
    }
}

struct face_groups_t {
    vector<double> xs, ys;
    vector<int> offset;
    vector<int4> faces;
};

auto get_faces_in_cells(device_vector<double3> &verts, device_vector<int4> &faces, grid_t &grid, int dir) {
    auto &xv = dir == 0 ? grid.ys : dir == 1 ? grid.zs : grid.xs;
    auto &yv = dir == 0 ? grid.zs : dir == 1 ? grid.xs : grid.ys;
    face_groups_t ret { xv, yv };

    device_vector xs(xv), ys(yv);
    ret.offset.resize(xs.len * ys.len + 1);
    device_vector<int> offset(ret.offset);
    kernel_get_faces_in_cells CU_DIM(1024, 128) (verts, faces, xs, ys, dir, offset, { });
    CUDA_ASSERT(hipGetLastError());

    offset.copy_to(ret.offset);
    auto sum = accumulate(ret.offset.begin(), ret.offset.end(), 0);
    device_vector<int4> out(sum);
    auto offset_vec = ret.offset;
    exclusive_scan(offset_vec.begin(), offset_vec.end(), ret.offset.begin(), 0);

    offset.copy_from(ret.offset);
    kernel_get_faces_in_cells CU_DIM(1024, 128) (verts, faces, xs, ys, dir, offset, out);
    CUDA_ASSERT(hipGetLastError());
    out.copy_to(ret.faces);
    return ret;
}

__host__ __device__ __forceinline__ auto lerp(double a, double b, double f) {
    return a * (1. - f) + b * f;
}
__host__ __device__ __forceinline__ auto interp(double2 a, double2 b, double x) {
    auto f = (x - a.x) / (b.x - a.x);
    return lerp(a.y, b.y, f);
}
__host__ __device__ __forceinline__ auto interp(double3 a, double3 b, double x) {
    auto f = (x - a.x) / (b.x - a.x);
    return double2 { lerp(a.y, b.y, f), lerp(a.z, b.z, f) };
}
__host__ __device__ __forceinline__ auto ordered(double x0, double x, double x1) {
    return (x0 < x1 && x0 <= x && x <= x1) || (x1 < x0 && x1 <= x && x <= x0);
}
__host__ __device__ __forceinline__ auto point_in_triangle(double2 p, double3 A, double3 B, double3 C, int dir) {
    double2 u = { 0 }, v = { 0 };
    auto a = rotate(A, dir), b = rotate(B, dir), c = rotate(C, dir);
    auto AB = ordered(a.x, p.x, b.x),
         BC = ordered(b.x, p.x, c.x),
         CA = ordered(c.x, p.x, a.x);
    if (CA && AB) {
        u = interp(a, b, p.x); v = interp(a, c, p.x);
    } else if (AB && BC) {
        u = interp(b, c, p.x); v = interp(b, a, p.x);
    } else if (BC && CA) {
        u = interp(c, a, p.x); v = interp(c, b, p.x);
    }
    if (ordered(u.x, p.y, v.x)) {
        return interp(u, v, p.y);
    }
    return DBL_MAX;
}

struct cast_joint_t {
    int solid;
    double pos;
};
auto operator<(const cast_joint_t &a, const cast_joint_t &b) {
    return a.solid == b.solid ? a.pos < b.pos : a.solid < b.solid;
}
__global__ void kernel_cast_in_cells(
    buffer<double3> verts, buffer<int4> faces,
    buffer<double> xs, buffer<double> ys, int dir,
    buffer<int> offset, double tol,
    buffer<int> len, buffer<cast_joint_t> out) {
    int u = blockIdx.x, v = blockIdx.y, w = u + gridDim.x * v;
    if (u >= xs.len - 1 || v >= ys.len - 1) {
        return;
    }
    int i = threadIdx.x, j = threadIdx.y, k = i + blockDim.x * j;
    double2 p = {
        lerp(xs[u] + tol, xs[u + 1] - tol, 1. * i / (blockDim.x - 1)),
        lerp(ys[v] + tol, ys[v + 1] - tol, 1. * j / (blockDim.y - 1)),
    };
    for (int m0 = offset[w], m1 = offset[w + 1]; m0 < m1; m0 ++) {
        auto face = faces[m0];
        auto pos = point_in_triangle(p, verts[face.x], verts[face.y], verts[face.z], dir);
        if (pos != DBL_MAX) {
            auto next = atomicAdd(len.ptr + w * blockDim.x * blockDim.y + k, 1);
            if (next < out.len) {
                out[next] = { face.w, pos };
            }
        }
    }
}

struct cast_dexel_t {
    double tol;
    int pixels;
    vector<double> xs, ys;
    vector<int> offset;
    vector<cast_joint_t> joints;
    auto sort_joints() {
        auto ptr = joints.data();
        for (int m = 0, n = offset.size(); m < n - 1; m ++) {
            auto begin = offset[m], end = offset[m + 1];
            sort(ptr + begin, ptr + end);
        }
    }
    auto dump_gltf(string file) {
        vector<double3> bin;
        for (int u = 0; u + 1 < xs.size(); u ++) for (int v = 0; v + 1 < ys.size(); v ++) {
            int w = u + v * xs.size();
            for (int i = 0; i < pixels; i ++) for (int j = 0; j < pixels; j ++) {
                int k = i + j * pixels;
                double2 p = {
                    lerp(xs[u] + tol, xs[u + 1] - tol, 1. * i / (pixels - 1)),
                    lerp(ys[v] + tol, ys[v + 1] - tol, 1. * j / (pixels - 1)),
                };
                auto m = w * pixels * pixels + k;
                auto begin = offset[m], end = offset[m + 1];
                if (0) {
                    for (int q = begin; q < end; q ++) {
                        auto &a = joints[q];
                        bin.push_back({ p.x, p.y, a.pos + tol * 10 });
                        bin.push_back({ p.x, p.y, a.pos - tol * 10 });
                    }
                } else {
                    for (int q = begin; q < end - 1; q ++) {
                        auto &a = joints[q], &b = joints[q + 1];
                        if (a.solid == b.solid) {
                            bin.push_back({ p.x, p.y, a.pos + tol });
                            bin.push_back({ p.x, p.y, b.pos - tol });
                            q ++;
                        }
                    }
                }
            }
        }

        vector<float3> out;
        for (auto p : bin) {
            out.push_back({ (float) p.x, (float) p.y, (float) p.z });
        }
        float3 p0 = { FLT_MAX, FLT_MAX, FLT_MAX }, p1 = { FLT_MIN, FLT_MIN, FLT_MIN };
        if (out.size()) {
            p0 = out.back(); p1 = out.front();
            for (auto p : out) {
                p0.x = fmin(p0.x, p.x); p1.x = fmax(p1.x, p.x);
                p0.y = fmin(p0.y, p.y); p1.y = fmax(p1.y, p.y);
                p0.z = fmin(p0.z, p.z); p1.z = fmax(p1.z, p.z);
            }
            auto d = float3 { p1.x - p0.x, p1.y - p0.y, p1.z - p0.z };
            out.push_back({ 0, 0, p0.z -= d.z });
            out.push_back({ 0, 0, p1.z += d.z });
            out.push_back({ 0, p0.y -= d.y, 0 });
            out.push_back({ 0, p1.y += d.z, 0 });
            out.push_back({ p0.x -= d.x, 0, 0 });
            out.push_back({ p1.x += d.x, 0, 0 });
        }
        std::ofstream fn(file + ".bin", std::ios::out | std::ios::binary);
        auto byteLength = out.size() * sizeof(float3);
        fn.write((char *) out.data(), byteLength);

        json j;
        std::ifstream("tool/view.gltf") >> j;
        for (auto &item : j["accessors"]) {
            item["min"][0] = p0.x; item["min"][1] = p0.y; item["min"][2] = p0.z;
            item["max"][0] = p1.x; item["max"][1] = p1.y; item["max"][2] = p1.z;
            item["count"] = out.size();
        }
        for (auto &item : j["bufferViews"]) {
            item["byteLength"] = byteLength;
        }
        auto filename = filesystem::path(file).filename().u8string();
        for (auto &item : j["buffers"]) {
            item["byteLength"] = byteLength;
            item["uri"] = filename + ".bin";
        }
        std::ofstream(file) << j.dump(2);
    }
};
auto cast_in_cells(device_vector<double3> &verts, face_groups_t &groups, int dir, int pixels, double tol) {
    device_vector xs(groups.xs), ys(groups.ys);
    device_vector faces(groups.faces);
    device_vector offset(groups.offset);
    dim3 gridDim((int) xs.len, (int) ys.len, 1),
         blockDim(pixels, pixels, 1);

    cast_dexel_t ret { tol, pixels, groups.xs, groups.ys };
    ret.offset.resize(xs.len * ys.len * blockDim.x * blockDim.y + 1);
    device_vector len(ret.offset);
    kernel_cast_in_cells CU_DIM(gridDim, blockDim) (verts, faces, xs, ys, dir, offset, tol, len, { });
    CUDA_ASSERT(hipGetLastError());

    len.copy_to(ret.offset);
    auto sum = accumulate(ret.offset.begin(), ret.offset.end(), 0);
    device_vector<cast_joint_t> out(sum);
    auto offset_vec = ret.offset;
    exclusive_scan(offset_vec.begin(), offset_vec.end(), ret.offset.begin(), 0);

    len.copy_from(ret.offset);
    kernel_cast_in_cells CU_DIM(gridDim, blockDim) (verts, faces, xs, ys, dir, offset, tol, len, out);
    CUDA_ASSERT(hipGetLastError());
    out.copy_to(ret.joints);
    ret.sort_joints();

    return ret;
}

int main() {
    grid_t grid;
    grid.from_json("data\\MaterialAndGridLines.json");
    grid.xs = add_midpoint(grid.xs); //grid.xs = add_midpoint(grid.xs);
    grid.ys = add_midpoint(grid.ys); //grid.ys = add_midpoint(grid.ys);
    grid.zs = add_midpoint(grid.zs); //grid.zs = add_midpoint(grid.zs);
    printf("loaded %zu x %zu x %zu (%zu) grids\n",
        grid.xs.size(), grid.ys.size(), grid.zs.size(),
        grid.xs.size() * grid.ys.size() * grid.zs.size());

    mesh_t mesh;
    mesh.from_obj("data\\toStudent_EM.obj");
    printf("loaded %zu faces with %zu vertices\n", mesh.faces.size(), mesh.verts.size());

    if (0) {
        grid.xs = { -2, -1, 0, 1, 2 };
        grid.ys = { -2, -1, 0, 1, 2 };
        grid.zs = { -2, -1, 0, 1, 2 };
        mesh.verts = {
            { 0, 0, 0 },
            { 0.5, 0, 0 },
            { 0, 0.7, 0 },
            { 0, 0, 0.9 },
        };
        mesh.faces = {
            { 0, 1, 2, 0 },
            { 0, 1, 3, 0 },
            { 0, 2, 3, 0 },
            { 1, 2, 3, 0 },
        };
    }

    device_vector verts(mesh.verts);
    device_vector faces(mesh.faces);
    auto all_start = clock_now();
    for (int dir = 0; dir < 3; dir ++) {
        auto start_group = clock_now();
        auto groups = get_faces_in_cells(verts, faces, grid, dir);
        printf("group %zu in %f s\n", groups.faces.size(), seconds_since(start_group));
        auto start_cast = clock_now();
        auto casted = cast_in_cells(verts, groups, dir, 32, 1e-3);
        printf("cast %zu in %f s\n", casted.joints.size(), seconds_since(start_cast));
        //casted.dump_gltf("data/dump-" + to_string(dir) + ".gltf");
    }
    printf("all done in %f s\n", seconds_since(all_start));

    return 0;
}
